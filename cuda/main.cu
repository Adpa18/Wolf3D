#include "hip/hip_runtime.h"
/*
** main.c for main in /home/wery_a/rendu/minilibX/FDF
**
** Made by adrien wery
** Login   <wery_a@epitech.net>
**
** Started on  Wed Nov 26 11:30:39 2014 adrien wery
** Last update Sun Dec 21 00:22:39 2014 adrien wery
*/

#include        "system.h"

__global__ void		init_vecs_cuda(t_vision *vecs, t_player *p, char *map)
{
    int		i;
    double	x1;
    double	y1;
    double cos_angle;
    double sin_angle;

    i = 0;
    x1 = p->pres / 2;
    if (p->angle / 180 * M_PI < 0) {
        p->angle = (360 + p->angle / 180 * M_PI) / M_PI * 180;
    }
    cos_angle = cos(p->angle);
    sin_angle = sin(p->angle);
    i = blockIdx.x;
    y1 = p->pres * ((HEIGHT / 2) - i) / HEIGHT;
    vecs[i].x1 = fabs(x1 * cos_angle - y1 * sin_angle + p->x0);
    vecs[i].y1 = fabs(x1 * sin_angle + y1 * cos_angle + p->y0);
    vecs[i].k = 0.001;
    vecs[i].x = fabs(p->x0 + vecs[i].k * (vecs[i].x1 - p->x0));
    vecs[i].y = fabs(p->y0 + vecs[i].k * (vecs[i].y1 - p->y0));
    while (map[vecs[i].x + vecs[i].y * p->len_x] != '1')
    {
        vecs[i].x = fabs(p->x0 + vecs[i].k * (vecs[i].x1 - p->x0));
        vecs[i].y = fabs(p->y0 + vecs[i].k * (vecs[i].y1 - p->y0));
        vecs[i].k += 0.001;
    }
}

int		       main()
{
    t_system    sys;
    int         fd;

    init_mlx(&sys);
    sys.player = init_player();
    sys.move = init_move();
    if ((fd = open("./laby", O_RDONLY)) == 1) {
        my_error("Error to open the map", 1);
    }
    check_map(open("./laby", O_RDONLY), sys.player);
    sys.map = wolf_tab(fd, (sys.player->len_x + 1) * (sys.player->len_y + 1));
    sys.vecs = (t_vision *)malloc(sizeof(t_vision) * WIDTH);

    t_vision *vecs;
    t_player *player;
    char *map;

    hipMalloc((void **)&vecs, sizeof(t_vision) * WIDTH);
    hipMalloc((void **)&player, sizeof(t_player));
    hipMalloc((void **)&map, (sys.player->len_x + 1) * (sys.player->len_y + 1));

    hipMemcpy(player, sys.player, sizeof(t_player), hipMemcpyHostToDevice);
    hipMemcpy(map, sys.map, (sys.player->len_x + 1) * (sys.player->len_y + 1), hipMemcpyHostToDevice);

    init_vecs_cuda<<<WIDTH,1>>>(vecs, player, map);

    hipMemcpy(sys.vecs, vecs, sizeof(t_vision) * WIDTH, hipMemcpyDeviceToHost);

    // int i = -1;
    // while (++i < WIDTH) {
    //     printf("k = %f\nx1 = %f\ny1 = %f\nx = %d\ny = %d\n", sys.vecs[i].k, sys.vecs[i].x1, sys.vecs[i].y1, sys.vecs[i].x, sys.vecs[i].y);
    // }
    my_hook(&sys);
    return (0);
}
